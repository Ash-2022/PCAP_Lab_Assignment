#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <math.h>

#define BLOCK_SIZE 16
#define LOW_THRESHOLD 50
#define HIGH_THRESHOLD 150

using namespace cv;

#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void gaussian_blur(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    float filter[5][5] = {
        {1, 4, 7, 4, 1},
        {4, 16, 26, 16, 4},
        {7, 26, 41, 26, 7},
        {4, 16, 26, 16, 4},
        {1, 4, 7, 4, 1}
    };
    
    float weight = 273.0;
    int radius = 2;
    float blur_value = 0.0;
    
    for (int i = -radius; i <= radius; i++) {
        for (int j = -radius; j <= radius; j++) {
            int nx = min(max(x + j, 0), width - 1);
            int ny = min(max(y + i, 0), height - 1);
            blur_value += input[ny * width + nx] * filter[i + radius][j + radius];
        }
    }
    output[y * width + x] = blur_value / weight;
}

__global__ void sobel_filter(unsigned char *input, unsigned char *output, float *gradient, float *direction, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int gx[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int gy[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};
    
    float grad_x = 0;
    float grad_y = 0;
    
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            int nx = min(max(x + j, 0), width - 1);
            int ny = min(max(y + i, 0), height - 1);
            grad_x += input[ny * width + nx] * gx[i + 1][j + 1];
            grad_y += input[ny * width + nx] * gy[i + 1][j + 1];
        }
    }
    gradient[y * width + x] = sqrt(grad_x * grad_x + grad_y * grad_y);
    direction[y * width + x] = atan2f(grad_y, grad_x);
    output[y * width + x] = min(255, (int)gradient[y * width + x]);
}

__global__ void non_maximum_suppression(float *gradient, float *direction, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1) {
        float angle = direction[y * width + x] * (180.0 / M_PI);
        angle = fmod(angle + 180.0, 180.0);
        
        float q = 255, r = 255;
        if ((angle >= 0 && angle < 22.5) || (angle >= 157.5 && angle <= 180)) {
            q = gradient[y * width + (x + 1)];
            r = gradient[y * width + (x - 1)];
        } else if (angle >= 22.5 && angle < 67.5) {
            q = gradient[(y + 1) * width + (x - 1)];
            r = gradient[(y - 1) * width + (x + 1)];
        } else if (angle >= 67.5 && angle < 112.5) {
            q = gradient[(y + 1) * width + x];
            r = gradient[(y - 1) * width + x];
        } else if (angle >= 112.5 && angle < 157.5) {
            q = gradient[(y - 1) * width + (x - 1)];
            r = gradient[(y + 1) * width + (x + 1)];
        }
        
        if (gradient[y * width + x] >= q && gradient[y * width + x] >= r) {
            output[y * width + x] = gradient[y * width + x];
        } else {
            output[y * width + x] = 0;
        }
    }
}

__global__ void double_threshold(float *gradient, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    float pixel = gradient[y * width + x];
    if (pixel >= HIGH_THRESHOLD) {
        output[y * width + x] = 255;
    } else if (pixel >= LOW_THRESHOLD) {
        output[y * width + x] = 128;
    } else {
        output[y * width + x] = 0;
    }
}

__global__ void edge_tracking_hysteresis(unsigned char *image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < 1 || y < 1 || x >= width - 1 || y >= height - 1) return;

    if (image[y * width + x] == 128) { // Weak edge
        if (image[(y - 1) * width + (x - 1)] == 255 || image[(y - 1) * width + x] == 255 ||
            image[(y - 1) * width + (x + 1)] == 255 || image[y * width + (x - 1)] == 255 ||
            image[y * width + (x + 1)] == 255 || image[(y + 1) * width + (x - 1)] == 255 ||
            image[(y + 1) * width + x] == 255 || image[(y + 1) * width + (x + 1)] == 255) {
            image[y * width + x] = 255;
        } else {
            image[y * width + x] = 0;
        }
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <image_path>\n", argv[0]);
        return -1;
    }

    Mat image = imread(argv[1], IMREAD_GRAYSCALE);
    if (image.empty()) {
        printf("Failed to load image\n");
        return -1;
    }

    int width = image.cols;
    int height = image.rows;
    size_t img_size = width * height * sizeof(unsigned char);
    size_t grad_size = width * height * sizeof(float);

    unsigned char *d_input, *d_blur, *d_sobel, *d_nms, *d_final;
    float *d_gradient, *d_direction;
    
    CHECK_CUDA_ERROR(hipMalloc(&d_input, img_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_blur, img_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_sobel, img_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_nms, img_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_final, img_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_gradient, grad_size));
    CHECK_CUDA_ERROR(hipMalloc(&d_direction, grad_size));
    
    CHECK_CUDA_ERROR(hipMemcpy(d_input, image.data, img_size, hipMemcpyHostToDevice));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    printf("Applying Gaussian Blur...\n");
    gaussian_blur<<<gridSize, blockSize>>>(d_input, d_blur, width, height);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    Mat blurred(height, width, CV_8UC1);
    CHECK_CUDA_ERROR(hipMemcpy(blurred.data, d_blur, img_size, hipMemcpyDeviceToHost));
    imwrite("gaussian_blur.png", blurred);
    
    printf("Applying Sobel Filter...\n");
    sobel_filter<<<gridSize, blockSize>>>(d_blur, d_sobel, d_gradient, d_direction, width, height);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    Mat sobel(height, width, CV_8UC1);
    CHECK_CUDA_ERROR(hipMemcpy(sobel.data, d_sobel, img_size, hipMemcpyDeviceToHost));
    imwrite("sobel.png", sobel);

    printf("Applying Non-Maximum Suppression...\n");
    non_maximum_suppression<<<gridSize, blockSize>>>(d_gradient, d_direction, d_nms, width, height);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    Mat nms(height, width, CV_8UC1);
    CHECK_CUDA_ERROR(hipMemcpy(nms.data, d_nms, img_size, hipMemcpyDeviceToHost));
    imwrite("nms.png", nms);
    
    printf("Applying Double Thresholding...\n");
    double_threshold<<<gridSize, blockSize>>>(d_gradient, d_final, width, height);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    Mat thresholded(height, width, CV_8UC1);
    CHECK_CUDA_ERROR(hipMemcpy(thresholded.data, d_final, img_size, hipMemcpyDeviceToHost));
    imwrite("double_threshold.png", thresholded);
    
    printf("Applying Edge Tracking with Hysteresis...\n");
    edge_tracking_hysteresis<<<gridSize, blockSize>>>(d_final, width, height);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    Mat hysteresis(height, width, CV_8UC1);
    CHECK_CUDA_ERROR(hipMemcpy(hysteresis.data, d_final, img_size, hipMemcpyDeviceToHost));
    imwrite("hysteresis.png", hysteresis);

    printf("Edge Detection Completed. Images saved.\n");

    hipFree(d_input);
    hipFree(d_blur);
    hipFree(d_sobel);
    hipFree(d_nms);
    hipFree(d_final);
    hipFree(d_gradient);
    hipFree(d_direction);

    return 0;
}
